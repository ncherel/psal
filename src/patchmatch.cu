#include "hip/hip_runtime.h"
#include "patchmatch.h"

#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <torch/torch.h>
#include <ATen/ATen.h>
#include <cmath>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename T, int PSZ>
__device__ T dist(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
		  at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
		  int32_t x1, int32_t y1,
		  int32_t x2, int32_t y2,
		  T cutoff=1e10);

template <typename T, int PSZ>
__global__ void initialise_shift_map(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
				     at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
				     at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
				     at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map,
				     at::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits> states);
template <typename T, int PSZ>
__global__ void propagation(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
			    at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
			    at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
			    at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map);

template <typename T, int PSZ>
__global__ void random_search(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
			      at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map,
			      at::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits> states);

template <typename T, int PSZ>
__global__ void backward_kernel(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
				at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
				at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> grad_t1,
				at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> grad_t2,
				at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
				at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map);

std::vector<at::Tensor> patchmatch_cuda(const at::Tensor t1,
					const at::Tensor t2,
					int patch_size=3,
					int n_iters=10) {
  auto H = t1.size(1);
  auto W = t1.size(2);
  auto shift_map = torch::full({ 2, K, H, W }, -1, t1.options().dtype(torch::kInt32));
  auto cost_map = torch::full({ K, H, W }, 0.0, t1.options());

  // Must make the grid large enough to cover all pixels
  const dim3 blocks(4, 4);
  const dim3 grid(int(t1.size(1) / blocks.x) + 1, int(t1.size(2) / blocks.y) + 1);

  // Initialise the random states
  auto states = torch::randint(2 << 16, { H, W }, t1.options().dtype(torch::kInt64));
  
  AT_DISPATCH_FLOATING_TYPES(t1.scalar_type(), "patchmatch_cuda", ([&] {
	auto shift_map_accessor = shift_map.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>();
	auto cost_map_accessor = cost_map.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
	auto t1_accessor = t1.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
	auto t2_accessor = t2.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
	auto states_accessor = states.packed_accessor32<int64_t, 2, torch::RestrictPtrTraits>();

	// Alternative to macro is the cpp index sequence but this requires a templated lambda
	// only possible with recent c++ and gcc versions
	// We keep this version for the time being
        #define patch_match_n(psize) { \
	  initialise_shift_map<scalar_t, psize><<<grid, blocks>>>(t1_accessor, t2_accessor, shift_map_accessor, cost_map_accessor, states_accessor); \
	  for(int i = 0; i < n_iters; i++) { \
	    propagation<scalar_t, psize><<<grid, blocks>>>(t1_accessor, t2_accessor, shift_map_accessor, cost_map_accessor); \
	    random_search<scalar_t, psize><<<grid, blocks>>>(t1_accessor, t2_accessor, shift_map_accessor, cost_map_accessor, states_accessor); \
          } }

	// Dispatch to template
	if (patch_size == 1) patch_match_n(1)
	if (patch_size == 3) patch_match_n(3)
	if (patch_size == 5) patch_match_n(5)
	if (patch_size == 7) patch_match_n(7)
	if (patch_size == 9) patch_match_n(9)
	if (patch_size == 11) patch_match_n(11)
	if (patch_size == 12) patch_match_n(13)
	if (patch_size == 15) patch_match_n(15)
	if (patch_size == 17) patch_match_n(17)
	if (patch_size == 19) patch_match_n(19)
	if (patch_size == 21) patch_match_n(21)
	if (patch_size == 23) patch_match_n(23)

  }));

  return {shift_map, cost_map};
}


std::vector<at::Tensor> backward_cuda(const at::Tensor t1,
				      const at::Tensor t2,
				      const at::Tensor shift_map,
				      const at::Tensor cost_map_grad,
				      int patch_size=3) {
  auto grad_t1 = torch::zeros_like(t1);
  auto grad_t2 = torch::zeros_like(t2);

  // Must make the grid large enough to cover all pixels
  const dim3 blocks(4, 4);
  const dim3 grid(int(t1.size(1) / blocks.x) + 1, int(t1.size(2) / blocks.y) + 1);

  AT_DISPATCH_FLOATING_TYPES(t1.scalar_type(), "backward_cuda", ([&] {
	auto t1_accessor = t1.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
	auto t2_accessor = t2.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
	auto grad_t1_accessor = grad_t1.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
	auto grad_t2_accessor = grad_t2.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
	auto shift_map_accessor = shift_map.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>();
	auto cost_map_accessor = cost_map_grad.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();

        #define backward_n(psize) {backward_kernel<scalar_t, psize><<<grid, blocks>>>(t1_accessor, t2_accessor, grad_t1_accessor, grad_t2_accessor, shift_map_accessor, cost_map_accessor);}

	// Dispatch to template
	if (patch_size == 1) backward_n(1)
	if (patch_size == 3) backward_n(3)
	if (patch_size == 5) backward_n(5)
	if (patch_size == 7) backward_n(7)
	if (patch_size == 9) backward_n(9)
	if (patch_size == 11) backward_n(11)
	if (patch_size == 12) backward_n(13)
	if (patch_size == 15) backward_n(15)
	if (patch_size == 17) backward_n(17)
	if (patch_size == 19) backward_n(19)
	if (patch_size == 21) backward_n(21)
	if (patch_size == 23) backward_n(23)
  }));

  return {grad_t1, grad_t2};
}



template <typename T, int PSZ>
__global__ void initialise_shift_map(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
				     at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
				     at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
				     at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map,
				     at::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits> states) {
  int32_t i = blockDim.x * blockIdx.x + threadIdx.x;
  int32_t j = blockDim.y * blockIdx.y + threadIdx.y;

  if (is_in_inner_boundaries<T, 3, PSZ>(t1, i, j)) {
    auto local_state = states[i][j];
    T local_heap[K];
    int32_t local_heap_shift[K*2];

    for(int k=0; k < K; k++) {
      auto ii = randint(PSZ/2, t2.size(1) - PSZ/2, &local_state);
      auto jj = randint(PSZ/2, t2.size(2) - PSZ/2, &local_state);

      while (!is_valid_match<T, PSZ>(t2, ii, jj)) {
	ii = randint(PSZ/2, t2.size(1) - PSZ/2, &local_state);
	jj = randint(PSZ/2, t2.size(2) - PSZ/2, &local_state);
      }

      auto distance = dist<T,PSZ>(t1, t2, i, j, ii, jj);
      add_to_heap(distance, local_heap, local_heap_shift, ii, jj, k+1);
    }

    // Write to memory
    for(int k=0; k < K; k++) {
      shift_map[0][k][i][j] = local_heap_shift[2*k];
      shift_map[1][k][i][j] = local_heap_shift[2*k+1];
      cost_map[k][i][j] = local_heap[k];
    }
    states[i][j] = local_state;
  }
  else if(i < t1.size(1) && j < t1.size(2)) {
    // For pixel at the borders, draw random shifts but do not compute
    // the distances as they are invalid
    auto local_state = states[i][j];
    for(int k=0; k < K; k++) {
      shift_map[0][k][i][j] = randint(PSZ/2, t2.size(1) - PSZ/2, &local_state);
      shift_map[1][k][i][j] = randint(PSZ/2, t2.size(2) - PSZ/2, &local_state);
      cost_map[k][i][j] = 0.0;
    }
    states[i][j] = local_state;
  }
}


/*
  Compute the distance between patches
 */
template <typename T, int PSZ>
__device__ T dist(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
		  at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
		  int32_t x1, int32_t y1,
		  int32_t x2, int32_t y2,
		  T cutoff) {
  T dist = 0.0;
  for(int c=0; c < t1.size(0); c++) {
    for(int i = -PSZ/2; i < PSZ/2 + 1; i++) {
      for(int j = -PSZ/2; j < PSZ/2 + 1; j++) {
	auto diff = t1[c][x1+i][y1+j] - t2[c][x2+i][y2+j];
	dist += diff * diff;
      }

      // Early return if already worse than current
      if (dist > cutoff) {
	return dist;
      }
    }
  }
  return dist;
}


template <typename T, int PSZ>
__global__ void propagation(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
			    at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
			    at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
			    at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map) {
  int32_t i = blockDim.x * blockIdx.x + threadIdx.x;
  int32_t j = blockDim.y * blockIdx.y + threadIdx.y;

  const int dis[4] = { 0, 1, 0, -1 };
  const int djs[4] = { 1, 0, -1, 0 };

  if(is_in_inner_boundaries<T, 3, PSZ>(t1, i, j)) {
    T local_heap[K];
    int local_shift[2*K];
    
    // Read heap from global memory
    for(int k=0; k < K; k++) {
      local_heap[k] = cost_map[k][i][j];
      local_shift[2*k] = shift_map[0][k][i][j];
      local_shift[2*k+1] = shift_map[1][k][i][j];
    }

    auto worst_distance = local_heap[0];

    for(int step_length=1; step_length <= 8; step_length *= 2) {
      for(int index = 0; index < 4; index++) {
	auto di = dis[index] * step_length;
	auto dj = djs[index] * step_length;

	if (!is_in_inner_boundaries<T, 3, PSZ>(t1, i+di, j+dj)) {
	  continue;
	}

	auto ii = shift_map[0][K-1][i+di][j+dj] - di;
	auto jj = shift_map[1][K-1][i+di][j+dj] - dj;
      
	if(is_valid_match<T, PSZ>(t2, ii, jj) && !in_heap(local_shift, ii, jj)) {
	  auto distance = dist<T, PSZ>(t1, t2, i, j, ii, jj, worst_distance);
	  if(distance < worst_distance) {
	    insert_into_heap(local_heap, local_shift, distance, ii, jj);
	    worst_distance = local_heap[0];
	  }
	}
      }
    }

    // Write to memory
    for(int k=0; k < K; k++) {
      shift_map[0][k][i][j] = local_shift[2*k];
      shift_map[1][k][i][j] = local_shift[2*k+1];
      cost_map[k][i][j] = local_heap[k];
    }
  }
}

template <typename T, int PSZ>
__global__ void random_search(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
			      at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map,
			      at::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits> states) {
  int32_t i = blockDim.x * blockIdx.x + threadIdx.x;
  int32_t j = blockDim.y * blockIdx.y + threadIdx.y;
  
  if(is_in_inner_boundaries<T, 3, PSZ>(t1, i, j)) {
    T local_heap[K];
    int local_shift[2*K];
    
    // Read heap from global memory
    for(int k=0; k < K; k++) {
      local_heap[k] = cost_map[k][i][j];
      local_shift[2*k] = shift_map[0][k][i][j];
      local_shift[2*k+1] = shift_map[1][k][i][j];
    }

    auto local_state = states[i][j];

    // Sample around current best
    auto best_ii = local_shift[2*(K-1)];
    auto best_jj = local_shift[2*(K-1)+1];

    // Worst match
    auto worst_distance = local_heap[0];

    const auto alpha = 0.5;
    auto wmax = max(t2.size(1), t2.size(2));
    int zmax = - logf(wmax) / logf(alpha);

    // Sample around the current match with a uniform window
    for(int z=0; z < zmax; z++) {
      int w = wmax * powf(alpha, z);
      int ii = randint(best_ii - w, best_ii + w, &local_state);
      int jj = randint(best_jj - w, best_jj + w, &local_state);
      
      if(is_valid_match<T,PSZ>(t2, ii, jj) && !in_heap(local_shift, ii, jj)) {
	auto distance = dist<T,PSZ>(t1, t2, i, j, ii, jj, worst_distance);
	if (distance < worst_distance) {
	  insert_into_heap(local_heap, local_shift, distance, ii, jj);
	  worst_distance = local_heap[0];
	}
      }
    }

    // Write to memory
    for(int k=0; k < K; k++) {
      shift_map[0][k][i][j] = local_shift[2*k];
      shift_map[1][k][i][j] = local_shift[2*k+1];
      cost_map[k][i][j] = local_heap[k];
    }
    states[i][j] = local_state;
  }
}

template <typename T, int PSZ>
__global__ void backward_kernel(at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t1,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> t2,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> grad_t1,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> grad_t2,
			      at::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits> shift_map,
			      at::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> cost_map) {
  int32_t i = blockDim.x * blockIdx.x + threadIdx.x;
  int32_t j = blockDim.y * blockIdx.y + threadIdx.y;

  // Compute the gradients using the patches that contain the given pixel (to avoid race conditions)
  if(is_in_inner_boundaries<T,3,PSZ>(t1, i, j)) {
    for(int k=0; k < K; k++) {
      for(int di=-PSZ/2; di < PSZ/2 + 1; di++) {
	for(int dj=-PSZ/2; dj < PSZ/2 + 1; dj++) {
	  // Shifted positions
	  auto ii = shift_map[0][k][i+di][j+dj];
	  auto jj = shift_map[1][k][i+di][j+dj];

	  for(int c=0; c < t1.size(0); c++) {
	    grad_t1[c][i][j] += 2 * (t1[c][i][j] - t2[c][ii-di][jj-dj]) * cost_map[k][i+di][j+dj];

	    // Race condition for grad_t2
	    grad_t2[c][ii-di][jj-dj] += 2 * (t2[c][ii-di][jj-dj] - t1[c][i][j]) * cost_map[k][i+di][j+dj];
	  }
	}
      }
    }
  }
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("patchmatch", &patchmatch_cuda, "PatchMatch implementation",
	pybind11::arg("t1"),
	pybind11::arg("t2"),
	pybind11::arg("patch_size") = 3,
	pybind11::arg("n_iters") = 10);
  m.def("backward", &backward_cuda, "Backward implementation",
	pybind11::arg("a"),
	pybind11::arg("b"),
	pybind11::arg("shift_map"),
	pybind11::arg("cost_map_grad"),
	pybind11::arg("patch_size") = 3);
}
